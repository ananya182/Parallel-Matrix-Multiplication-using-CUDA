
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <chrono>

#define ll unsigned long long 

using namespace std;

__global__ void multiply(short *blocks1, short *blocks2, int *bidx2, int *bidy1, int *strow1, int *strow2, int *endrow1, int *endrow2, unsigned int *C, int n, int m)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * m + ty;
    int col = bx * m + tx;
    ll temp = 0,maxval=4294967295;

    int a=strow1[row],b=endrow1[row];
    int p=strow2[col],q=endrow2[col];

    if(a!=-1 && p!=-1){

    while(a<=b && p<=q){

        if (bidy1[a] == bidx2[p]){

        for (int k = 0; k < m; ++k)
        {
        
            temp += blocks1[m*m*a+m*ty+k] * blocks2[m*m*p+m*k+tx];
        }
        a++;p++;
        
        }

        else if(bidy1[a] < bidx2[p])
        a++;

        else
        p++;
        
    }}
    if(temp>maxval)
    C[row * n + col] = maxval;
    else
    C[row * n + col] =temp;

}

int main(int argc, char **argv)
{
    string infile1 = argv[1];
    string infile2 = argv[2];
    string outfile = argv[3];

    ifstream f1(infile1, ios::binary);
    ifstream f2(infile2, ios::binary);

    int n1, m1, k1, n2, m2, k2, i, j;
    short x;

    f1.read((char *)(&n1), 4);
    f1.read((char *)(&m1), 4);
    f1.read((char *)(&k1), 4);

    vector<short>blocks1,blocks2;
    vector<int>blockidx,blockidy,row_wise_startindex(n1,-1),row_wise_endindex(n1,-1),blockidx2,blockidy2,row_wise_startindex2(n1,-1),row_wise_endindex2(n1,-1);

    short *v1;

    v1 = (short *)malloc(n1 * n1 * sizeof(short));

    for (int p = 0; p < n1 * n1; p++)
    {
        v1[p] = -1;
    }

    for (int p = 0; p < k1; p++)
    {
        f1.read((char *)(&i), 4);
        f1.read((char *)(&j), 4);

        for (int q = 0; q < m1 * m1; q++)
        {
            f1.read((char *)(&x), 2);
            v1[n1 * (i * m1 + q / m1) + j * m1 + q % m1] = x;
        }
    }

    for(int i=0;i<n1/m1;i++){
        for(int j=0;j<n1/m1;j++){
            if(v1[n1 *i * m1 + j * m1]==-1)
            continue;
            else{
            blockidx.push_back(i);
            blockidy.push_back(j);

            }

            for(int q=0;q<m1*m1;q++){
            blocks1.push_back(v1[n1 * (i * m1 + q / m1) + j * m1 + q % m1]);
        }
    }}
    
    int startrow=blockidx[0]*m1,endrow=startrow+m1-1,cur=blockidx[0];

    for(int i=startrow;i<=endrow;i++){
        row_wise_startindex[i]=0;

    }

    for(int i=0;i<blockidx.size();i++){
        if(blockidx[i]!=cur){
        for(int j=startrow;j<=endrow;j++){
        row_wise_endindex[j]=i-1;
    }
    cur=blockidx[i];
    startrow=cur*m1;
    endrow=startrow+m1-1;
    for(int j=startrow;j<=endrow;j++){
        row_wise_startindex[j]=i;
    }}}
        
    for(int j=startrow;j<=endrow;j++){
        row_wise_endindex[j]=blockidx.size()-1;
    }

    f2.read((char *)(&n2), 4);
    f2.read((char *)(&m2), 4);
    f2.read((char *)(&k2), 4);

    for (int p = 0; p < n1 * n1; p++)
    {
        v1[p] = -1;
    }

   for (int p = 0; p < k2; p++)
    {
        f2.read((char *)(&i), 4);
        f2.read((char *)(&j), 4);

        for (int q = 0; q < m1 * m1; q++)
        {
            f2.read((char *)(&x), 2);
            v1[n1 * (i * m1 + q / m1) + j * m1 + q % m1] = x;
        }
    }

    for(int j=0;j<n1/m1;j++){
        for(int i=0;i<n1/m1;i++){
            if(v1[n1 *i * m1 + j * m1]==-1)
            continue;
            else{
            blockidx2.push_back(i);
            blockidy2.push_back(j);

            }

            for(int q=0;q<m1*m1;q++){
            blocks2.push_back(v1[n1 * (i * m1 + q / m1) + j * m1 + q % m1]);
        }
    }}
    
    startrow=blockidy2[0]*m1,endrow=startrow+m1-1,cur=blockidy2[0];

    for(int i=startrow;i<=endrow;i++){
        row_wise_startindex2[i]=0;

    }

    for(int i=0;i<blockidx2.size();i++){
        if(blockidy2[i]!=cur){
        for(int j=startrow;j<=endrow;j++){
        row_wise_endindex2[j]=i-1;
    }
    cur=blockidy2[i];
    startrow=cur*m1;
    endrow=startrow+m1-1;
    for(int j=startrow;j<=endrow;j++){
        row_wise_startindex2[j]=i;
    }}}
        
    for(int j=startrow;j<=endrow;j++){
        row_wise_endindex2[j]=blockidx2.size()-1;
    }

    short *deviceb1,*deviceb2;
    int *bidy1,*bidx2,*strow1,*endrow1,*strow2,*endrow2;
    unsigned int *output, *deviceoutput;

    hipMalloc((void **)&deviceb1, blocks1.size() * sizeof(short));
    hipMalloc((void **)&deviceb2, blocks2.size() * sizeof(short));
    hipMalloc((void **)&bidx2, blockidx2.size() * sizeof(int));
    hipMalloc((void **)&bidy1, blockidy.size() * sizeof(int));
    hipMalloc((void **)&strow1, row_wise_startindex.size() * sizeof(int));
    hipMalloc((void **)&strow2, row_wise_startindex2.size() * sizeof(int));
    hipMalloc((void **)&endrow1, row_wise_endindex.size() * sizeof(int));
    hipMalloc((void **)&endrow2, row_wise_endindex2.size() * sizeof(int));

    hipMemcpy(deviceb1, blocks1.data(), blocks1.size() * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(deviceb2, blocks2.data(), blocks2.size() * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(bidx2, blockidx2.data(), blockidx2.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(bidy1, blockidy.data(), blockidy.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(strow1, row_wise_startindex.data(), row_wise_startindex.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(strow2, row_wise_startindex2.data(), row_wise_startindex2.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(endrow1, row_wise_endindex.data(), row_wise_endindex.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(endrow2, row_wise_endindex2.data(), row_wise_endindex2.size() * sizeof(int), hipMemcpyHostToDevice);

    dim3 DimGrid(n1 / m1 , n1 / m1 , 1);
    dim3 DimBlock(m1, m1, 1);

    hipMalloc((void **)&deviceoutput, n2 * n2 * sizeof(unsigned int));
    multiply<<<DimGrid, DimBlock>>>(deviceb1, deviceb2, bidx2,bidy1,strow1,strow2,endrow1,endrow2, deviceoutput, n1, m1);
    hipDeviceSynchronize();

    output = (unsigned int *)malloc(n1 * n1 * sizeof(unsigned int));
    hipMemcpy(output, deviceoutput, n1 * n1 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(deviceb1);hipFree(deviceb2);hipFree(bidx2);hipFree(bidy1);
    hipFree(strow1);hipFree(strow2);hipFree(endrow1);hipFree(endrow2);
    hipFree(deviceoutput);
   
    ofstream out (outfile, ios_base::binary);
    vector<pair<int,int>>blocks;

    out.write((char *)(&n1), 4);
    out.write((char *)(&m1), 4);

    for(int i=0;i<n1/m1;i++){
        for(int j=0;j<n1/m1;j++){
            for(int q=0;q<m1*m1;q++){
            if(output[n2 * (i * m2 + q / m2) + j * m2 + q % m2] != 0){
            blocks.push_back({i,j});
            break;}
        }
    }}

    int k=blocks.size();
    out.write((char *)(&k), 4);

    for(int i=0;i<blocks.size();i++){
        out.write((char *)(&blocks[i].first), 4);
        out.write((char *)(&blocks[i].second), 4);

        for(int j=0;j<m1*m1;j++){
            out.write((char *)(&output[n1 * (blocks[i].first * m1 + j / m1) + blocks[i].second * m1 + j % m1]), 4);
        }
    }

    free(output);
}

   

  

  
   





    



